#include "hip/hip_runtime.h"
# define COUNT_OCCURENCES_V1

#include "hip/hip_runtime.h"
#include ""

#include <math.h>
#include <memory.h>
#include "cuda_errors.cuh"



/// <summary>
/// Calcola quanti blocchi si deve lanciare per processare 
/// un vettore di lungh n, con threadsPerBlock thread per blocco.
/// </summary>
/// <param name="n"></param>
/// <param name="threadsPerBlock"></param>
/// <returns></returns>
int _calculateB(int n, int threadsPerBlock) {
	return (int) ceil(
		(double)n /
		(double)threadsPerBlock
	);
}

/// <summary>
/// SOLUZIONE 1: 1 thread per dato, uso di atomicAdd
/// </summary>
/// <param name="v"></param>
/// <param name="n"></param>
/// <param name="x"></param>
/// <param name="result"></param>
/// <returns></returns>
__global__ void countOccurrenciesV1_kernel(int* v, int n, int x, int* result) {

	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid < n) {
		if (v[tid] == x) {
			atomicAdd(result, 1);
		}
	}
}

/// <summary>
/// Conta le occorrenze di x nel vettore v
/// </summary>
/// <param name="v"></param>
/// <param name="n"></param>
/// <param name="x"></param>
/// <return>Il numero di occorrenze osservate</return>
int countOccurrenciesV1(int* v, int n, int x) {

	// allocazione memoria su device
	// - n spazi per il vettore di input
	// - 1 spazio finale per il risultato
	int* vDevice;
	HANDLE_ERROR(hipMalloc((void**)&vDevice, sizeof(int) * (n + 1)));
	int* resultDevice = &vDevice[n];

	// copia vettore di input nel device + inizializzazione a 0 spazio risultato
	HANDLE_ERROR(hipMemcpy(vDevice, v, sizeof(int) * n, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemset(resultDevice, 0, sizeof(int)));

	// calcolo numero blocchi
	const int threadsPerBlock = 256;
	int nBlocks = _calculateB(n, threadsPerBlock);

	// lancio kernel
	countOccurrenciesV1_kernel << < nBlocks, threadsPerBlock >> > (vDevice, n, x, resultDevice);
	hipDeviceSynchronize();
	checkKernelError("kernel v1");

	// copia risultato su host
	int result;
	HANDLE_ERROR(hipMemcpy(&result, resultDevice, sizeof(int), hipMemcpyDeviceToHost));

	// liberazione memoria device
	HANDLE_ERROR(hipFree(vDevice));

	return result;
}