﻿#include <time.h>
#include <stdio.h>
#include <memory.h>
#include <stdlib.h>
#include <iostream>
#include "count_x_v1.cuh"
#include "count_x_v2.cuh"


/// <summary>
/// Conta le occorrenze di x nel vettore v
/// </summary>
/// <param name="v"></param>
/// <param name="n"></param>
/// <param name="x"></param>
/// <return>Il numero di occorrenze osservate</return>
int countOccurrenciesV0(int* v, int n, int x) {

	int xOccurrencies = 0;

	for (size_t i = 0; i < n; i++)
	{
		if (v[i] == x) {
			xOccurrencies++;
		}
	}

	return xOccurrencies;
}

/// <summary>
/// Riempi lo spazio v (di dimensione n) con valori casuali
/// appartenenti all'intervallo [startValue, endValue]
/// </summary>
/// <param name="v"></param>
/// <param name="n"></param>
/// <param name="startValue"></param>
/// <param name="endValue"></param>
void randomArray(int* v, int n, int startValue, int endValue) {

	time_t t;
	srand((unsigned)time(&t));

	for (size_t i = 0; i < n; i++)
	{
		v[i] = (rand() % (endValue - startValue)) + startValue;
	}
}

int main() {

	int n, start, end, x, nBlocks, threadsPerBlocks;
	bool defaultOptions = false;

	std::cout << "Inserisci la dimensione del vettore [-1 per usare opzioni di default]:\t";
	std::cin >> n;
	// n = 10000;
	if (n<=0) {
		n = 10000; start = 0; end = 10; x = 5; nBlocks = 8; threadsPerBlocks = 128;
		defaultOptions = true;

		std::cout << "Caricamento opzioni di default: vettore casuale di " << n
			<< " elementi, compresi in {" << start << ".." << end << "-1}. Conta occorrenze di "
			<< x << ". Quanto posso scegliere, uso " << nBlocks << " blocchi da "
			<< threadsPerBlocks << "thread ciascuno.";
	}
	else {
		std::cout << "\nInserisci l'intervallo dei valori del vettore.\nValore minimo:\t";
		std::cin >> start;
		// start = 0;
		std::cout << "Valore massimo:\t";
		std::cin >> end;
		// end = 100;

		std::cout << "\nInserisci valore da cercare:\t";
		std::cin >> x;
		// x = 0;

		std::cout << "\nInserisci numero di blocchi e thread per blocco da usare nella computazione parallela V2.\nNumero di Blocchi:\t";
		std::cin >> nBlocks;
		std::cout << "Thread per blocco:\t";
		std::cin >> threadsPerBlocks;
	}

	int* v = (int*)malloc(sizeof(int) * n);
	randomArray(v, n, start, end);

	printf("\nNumero di occorrenze (seriale):%i\t", countOccurrenciesV0(v, n, x));
	printf("\nNumero di occorrenze (parallelo v1):%i\t", countOccurrenciesV1(v, n, x));
	printf("\nNumero di occorrenze (parallelo v2):%i\t", countOccurrenciesV2(v, n, x, nBlocks, threadsPerBlocks, COALESCENCE));

	return 0;
}
